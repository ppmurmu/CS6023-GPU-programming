#include <iostream>
#include <fstream>
#include <vector>
#include <queue>
#include <algorithm>
#include <climits>
#include <cmath>
#include <unordered_map>
#include <utility>
#include <hip/hip_runtime.h>


using namespace std;

// Structure to represent a road between cities
struct Road
{
    int to;
    int length;
    int capacity;
};

// CUDA kernel for initializing distance matrix
__global__ void initializeDistancesKernel(int *d_distances, int num_cities)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_cities * num_cities)
    {
        int source = tid / num_cities;
        int dest = tid % num_cities;

        if (source == dest)
        {
            d_distances[tid] = 0; // Distance to self is 0
        }
        else
        {
            d_distances[tid] = INT_MAX; // Initially, all other distances are infinity
        }
    }
}

// Helper function for finding min distance vertex not in visited
__device__ int minDistance(int *dist, bool *visited, int num_cities)
{
    int min = INT_MAX, min_index = -1;

    for (int v = 0; v < num_cities; v++)
    {
        if (!visited[v] && dist[v] <= min)
        {
            min = dist[v];
            min_index = v;
        }
    }
    return min_index;
}

// CUDA kernel for Dijkstra's algorithm (multiple sources in parallel)
__global__ void parallelDijkstraKernel(int *d_graph, int *d_distances, int *d_paths, int num_cities)
{
    int source = blockIdx.x; // Each block handles one source vertex

    if (source >= num_cities)
        return;

    // Create a local distance array for this source
    int *dist = new int[num_cities];
    bool *visited = new bool[num_cities];

    // Initialize
    for (int i = 0; i < num_cities; i++)
    {
        dist[i] = INT_MAX;
        visited[i] = false;
    }
    dist[source] = 0;

    // Find shortest path for all vertices
    for (int count = 0; count < num_cities - 1; count++)
    {
        int u = minDistance(dist, visited, num_cities);

        if (u == -1)
            break; // No more reachable vertices

        visited[u] = true;

        // Update dist value of adjacent vertices
        for (int v = 0; v < num_cities; v++)
        {
            // Get edge weight from the flattened graph matrix
            int edge = d_graph[u * num_cities + v];

            // Update dist[v] if not visited, there is an edge, and path through u is shorter
            if (!visited[v] && edge != 0 && dist[u] != INT_MAX && dist[u] + edge < dist[v])
            {
                dist[v] = dist[u] + edge;
                d_paths[source * num_cities + v] = u; // predecessor for path reconstruction
            }
        }
    }

    // Copy results back to global memory
    for (int i = 0; i < num_cities; i++)
    {
        d_distances[source * num_cities + i] = dist[i];
    }

    delete[] dist;
    delete[] visited;
}

// Kernel for parallel path reconstruction
__global__ void pathReconstructionKernel(int *d_paths, int *d_reconstructed_paths, int *d_path_lengths, int num_cities, int max_path_length)
{
    int source = blockIdx.x;
    int dest = threadIdx.x;

    if (source >= num_cities || dest >= num_cities)
        return;

    // Skip self-paths (handled separately)
    if (source == dest)
    {
        d_path_lengths[source * num_cities + dest] = 1;
        d_reconstructed_paths[(source * num_cities + dest) * max_path_length] = source;
        return;
    }

    // Check if path exists
    if (d_paths[source * num_cities + dest] == -1)
    {
        d_path_lengths[source * num_cities + dest] = 0;
        return;
    }

    // Reconstruct path by following predecessors
    int pathIdx = 0;
    int current = dest;
    int temp_path[1000]; // Temp buffer for path (adjust size as needed)

    while (current != -1 && current != source)
    {
        temp_path[pathIdx++] = current;
        current = d_paths[source * num_cities + current];

        // Safety check for cycles
        if (pathIdx >= max_path_length - 1)
            break;
    }

    if (current == source)
    {
        temp_path[pathIdx++] = source;

        // Write path length
        d_path_lengths[source * num_cities + dest] = pathIdx;

        // Reverse and copy to output
        for (int i = 0; i < pathIdx; i++)
        {
            d_reconstructed_paths[(source * num_cities + dest) * max_path_length + i] = temp_path[pathIdx - 1 - i];
        }
    }
    else
    {
        // No valid path
        d_path_lengths[source * num_cities + dest] = 0;
    }
}

// Kernel for parallel conversion of adjacency list to matrix
__global__ void adjListToMatrixKernel(int *d_edges, int *d_matrix, int num_edges, int num_cities)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_edges)
    {
        int u = d_edges[tid * 4];
        int v = d_edges[tid * 4 + 1];
        int length = d_edges[tid * 4 + 2];

        // Set edge in matrix (undirected graph)
        d_matrix[u * num_cities + v] = length;
        d_matrix[v * num_cities + u] = length;
    }
}

// CUDA kernel for parallel shelter evaluation
__global__ void evaluateSheltersKernel(
    int *d_shelterCities,
    int *d_shelterCapacities,
    int *d_distances,
    float *d_scores,
    int sourceCity,
    int peopleToEvacuate,
    int num_shelters,
    int max_distance_elderly,
    bool forElderly,
    int num_cities)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= num_shelters)
        return;

    int shelterCity = d_shelterCities[tid];
    int capacity = d_shelterCapacities[tid];

    // Skip full shelters
    if (capacity <= 0)
    {
        d_scores[tid] = -1.0f;
        return;
    }

    int dist = d_distances[sourceCity * num_cities + shelterCity];

    // Skip if distance exceeds elderly limit and we're checking for elderly
    if (forElderly && dist > max_distance_elderly)
    {
        d_scores[tid] = -1.0f;
        return;
    }

    // Skip if no path exists
    if (dist == INT_MAX)
    {
        d_scores[tid] = -1.0f;
        return;
    }

    // Calculate people we can save
    int peopleSaved = min(peopleToEvacuate, capacity);

    // Score formula prioritizes people saved but also considers time
    float score = peopleSaved / (1.0f + 0.1f * dist);

    d_scores[tid] = score;
}

// Convert adjacency list to adjacency matrix for CUDA processing
void convertAdjListToMatrix(
    const vector<vector<Road>> &adjacencyList,
    int *matrix,
    int num_cities,
    int *roads,
    int num_roads)
{
    // Initialize all to 0 (no connection)
    memset(matrix, 0, num_cities * num_cities * sizeof(int));

    // Prepare to use GPU
    int *d_edges, *d_matrix;

    // Allocate device memory
    hipMalloc((void **)&d_edges, num_roads * 4 * sizeof(int));
    hipMalloc((void **)&d_matrix, num_cities * num_cities * sizeof(int));

    // Initialize matrix to 0
    hipMemset(d_matrix, 0, num_cities * num_cities * sizeof(int));

    // Copy edges to device
    hipMemcpy(d_edges, roads, num_roads * 4 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel for parallel matrix construction
    int blockSize = 256;
    int numBlocks = (num_roads + blockSize - 1) / blockSize;
    adjListToMatrixKernel<<<numBlocks, blockSize>>>(d_edges, d_matrix, num_roads, num_cities);

    // Copy result back to host
    hipMemcpy(matrix, d_matrix, num_cities * num_cities * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_edges);
    hipFree(d_matrix);
}

// Compute shortest paths using CUDA
void computeShortestPathsCuda(
    const vector<vector<Road>> &adjacencyList,
    vector<vector<int>> &distances,
    vector<vector<vector<int>>> &shortestPaths,
    int num_cities,
    int *roads,
    int num_roads)
{
    // Convert adjacency list to matrix for CUDA processing
    int *graphMatrix = new int[num_cities * num_cities];
    convertAdjListToMatrix(adjacencyList, graphMatrix, num_cities, roads, num_roads);

    // Create flattened arrays for distances and paths
    int *flatDistances = new int[num_cities * num_cities];
    int *flatPaths = new int[num_cities * num_cities];

    // Max path length (for path reconstruction)
    int max_path_length = num_cities;

    // Arrays for reconstructed paths
    int *flatReconstructedPaths = new int[num_cities * num_cities * max_path_length];
    int *pathLengths = new int[num_cities * num_cities];

    // Device memory
    int *d_graph, *d_distances, *d_paths, *d_reconstructed_paths, *d_path_lengths;

    // Allocate device memory
    hipMalloc((void **)&d_graph, num_cities * num_cities * sizeof(int));
    hipMalloc((void **)&d_distances, num_cities * num_cities * sizeof(int));
    hipMalloc((void **)&d_paths, num_cities * num_cities * sizeof(int));
    hipMalloc((void **)&d_reconstructed_paths, num_cities * num_cities * max_path_length * sizeof(int));
    hipMalloc((void **)&d_path_lengths, num_cities * num_cities * sizeof(int));

    // Copy graph to device
    hipMemcpy(d_graph, graphMatrix, num_cities * num_cities * sizeof(int), hipMemcpyHostToDevice);

    // Initialize distances
    int blockSize = 256;
    int numBlocks = (num_cities * num_cities + blockSize - 1) / blockSize;
    initializeDistancesKernel<<<numBlocks, blockSize>>>(d_distances, num_cities);

    // Initialize paths to -1 (no path)
    hipMemset(d_paths, -1, num_cities * num_cities * sizeof(int));

    // Launch Dijkstra kernel for all sources in parallel
    parallelDijkstraKernel<<<num_cities, 1>>>(d_graph, d_distances, d_paths, num_cities);

    // Synchronize to ensure Dijkstra is complete
    hipDeviceSynchronize();

    // Launch path reconstruction kernel
    pathReconstructionKernel<<<num_cities, num_cities>>>(d_paths, d_reconstructed_paths, d_path_lengths, num_cities, max_path_length);

    // Synchronize to ensure path reconstruction is complete
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(flatDistances, d_distances, num_cities * num_cities * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pathLengths, d_path_lengths, num_cities * num_cities * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(flatReconstructedPaths, d_reconstructed_paths, num_cities * num_cities * max_path_length * sizeof(int), hipMemcpyDeviceToHost);

    // Convert flat distances back to 2D vector
    for (int i = 0; i < num_cities; i++)
    {
        for (int j = 0; j < num_cities; j++)
        {
            distances[i][j] = flatDistances[i * num_cities + j];
        }
    }

    // Convert reconstructed paths to 3D vector format
    for (int source = 0; source < num_cities; source++)
    {
        for (int dest = 0; dest < num_cities; dest++)
        {
            int pathLength = pathLengths[source * num_cities + dest];

            if (pathLength > 0)
            {
                shortestPaths[source][dest].resize(pathLength);
                for (int k = 0; k < pathLength; k++)
                {
                    shortestPaths[source][dest][k] = flatReconstructedPaths[(source * num_cities + dest) * max_path_length + k];
                }
            }
            else
            {
                shortestPaths[source][dest].clear();
            }
        }
    }

    // Free device memory
    hipFree(d_graph);
    hipFree(d_distances);
    hipFree(d_paths);
    hipFree(d_reconstructed_paths);
    hipFree(d_path_lengths);

    // Free host memory
    delete[] graphMatrix;
    delete[] flatDistances;
    delete[] flatPaths;
    delete[] flatReconstructedPaths;
    delete[] pathLengths;
}

// Parallel shelter evaluation wrapper
vector<pair<int, double>> evaluateShelters(
    int sourceCity,
    int peopleToEvacuate,
    const vector<int> &shelterCities,
    const unordered_map<int, int> &shelterCapacity,
    const vector<vector<int>> &distances,
    int max_distance_elderly,
    bool forElderly = false)
{
    int num_shelters = shelterCities.size();
    if (num_shelters == 0)
    {
        return {};
    }

    // Prepare data for GPU
    int *d_shelterCities;
    int *d_shelterCapacities;
    int *d_distances;
    float *d_scores;

    int *h_shelterCities = new int[num_shelters];
    int *h_shelterCapacities = new int[num_shelters];
    float *h_scores = new float[num_shelters];

    // Flatten data
    int num_cities = distances.size();
    int *h_distancesFlat = new int[num_cities * num_cities];

    // Copy shelter cities and capacities
    for (int i = 0; i < num_shelters; i++)
    {
        h_shelterCities[i] = shelterCities[i];
        h_shelterCapacities[i] = shelterCapacity.at(shelterCities[i]);
    }

    // Flatten distance matrix
    for (int i = 0; i < num_cities; i++)
    {
        for (int j = 0; j < num_cities; j++)
        {
            h_distancesFlat[i * num_cities + j] = distances[i][j];
        }
    }

    // Allocate GPU memory
    hipMalloc((void **)&d_shelterCities, num_shelters * sizeof(int));
    hipMalloc((void **)&d_shelterCapacities, num_shelters * sizeof(int));
    hipMalloc((void **)&d_distances, num_cities * num_cities * sizeof(int));
    hipMalloc((void **)&d_scores, num_shelters * sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_shelterCities, h_shelterCities, num_shelters * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shelterCapacities, h_shelterCapacities, num_shelters * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_distances, h_distancesFlat, num_cities * num_cities * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (num_shelters + blockSize - 1) / blockSize;
    evaluateSheltersKernel<<<numBlocks, blockSize>>>(
        d_shelterCities, d_shelterCapacities, d_distances, d_scores,
        sourceCity, peopleToEvacuate, num_shelters, max_distance_elderly, forElderly, num_cities);

    // Get results
    hipMemcpy(h_scores, d_scores, num_shelters * sizeof(float), hipMemcpyDeviceToHost);

    // Create result vector
    vector<pair<int, double>> shelterScores;
    for (int i = 0; i < num_shelters; i++)
    {
        if (h_scores[i] > 0)
        {
            shelterScores.push_back({shelterCities[i], h_scores[i]});
        }
    }

    // Sort by score (highest first)
    sort(shelterScores.begin(), shelterScores.end(),
         [](const pair<int, double> &a, const pair<int, double> &b)
         {
             return a.second > b.second;
         });

    // Free memory
    delete[] h_shelterCities;
    delete[] h_shelterCapacities;
    delete[] h_scores;
    delete[] h_distancesFlat;

    hipFree(d_shelterCities);
    hipFree(d_shelterCapacities);
    hipFree(d_distances);
    hipFree(d_scores);

    return shelterScores;
}

// Generate evacuation paths for all populated cities
// Note: This function is kept sequential as it has complex dependencies
// that would be difficult to parallelize without breaking functionality
void generateEvacuationPaths(
    int num_cities,
    const vector<vector<Road>> &adjacencyList,
    const vector<int> &populatedCities,
    const unordered_map<int, pair<int, int>> &populatedCityInfo,
    const vector<int> &shelterCities,
    unordered_map<int, int> &shelterCapacity,
    const vector<vector<int>> &distances,
    const vector<vector<vector<int>>> &shortestPaths,
    int max_distance_elderly,
    long long *path_size,
    long long **paths,
    long long *num_drops,
    long long ***drops,
    int num_populated_cities)
{
    // For each populated city
    for (int i = 0; i < num_populated_cities; i++)
    {
        int sourceCity = populatedCities[i];
        int prime_age = populatedCityInfo.at(sourceCity).first;
        int elderly = populatedCityInfo.at(sourceCity).second;

        vector<int> evacPath;
        vector<vector<long long>> evacDrops;

        // Start with the source city
        evacPath.push_back(sourceCity);
        int currentCity = sourceCity;
        int distanceTraveled = 0;

        // Check if source city is a shelter
        if (shelterCapacity.find(sourceCity) != shelterCapacity.end() && shelterCapacity[sourceCity] > 0)
        {
            int capacity = shelterCapacity[sourceCity];

            // Drop elderly first
            int elderlyToDrop = min(elderly, capacity);
            capacity -= elderlyToDrop;
            elderly -= elderlyToDrop;

            // Then drop prime age
            int primeToDrop = min(prime_age, capacity);
            capacity -= primeToDrop;
            prime_age -= primeToDrop;

            // Update shelter capacity
            shelterCapacity[sourceCity] -= (elderlyToDrop + primeToDrop);

            // Record drop
            if (elderlyToDrop > 0 || primeToDrop > 0)
            {
                evacDrops.push_back({(long long)sourceCity, (long long)primeToDrop, (long long)elderlyToDrop});
            }
        }

        // Continue evacuation until all people are dropped
        while (prime_age > 0 || elderly > 0)
        {
            // First handle elderly (they have distance restrictions)
            int nextShelterForElderly = -1;
            auto elderlyShelters = evaluateShelters(
                currentCity, elderly, shelterCities, shelterCapacity,
                distances, max_distance_elderly, true);

            if (elderly > 0 && !elderlyShelters.empty())
            {
                nextShelterForElderly = elderlyShelters[0].first;
            }

            // If no shelter for elderly within range, drop them at current city
            if (elderly > 0 && nextShelterForElderly == -1)
            {
                evacDrops.push_back({(long long)currentCity, 0, (long long)elderly});
                elderly = 0;
            }

            // Now handle prime-age people
            int nextShelterForPrime = -1;
            auto primeShelters = evaluateShelters(
                currentCity, prime_age, shelterCities, shelterCapacity,
                distances, max_distance_elderly, false);

            if (prime_age > 0 && !primeShelters.empty())
            {
                nextShelterForPrime = primeShelters[0].first;
            }

            // If no shelter available for anyone, drop remaining at current city
            if (nextShelterForElderly == -1 && nextShelterForPrime == -1)
            {
                if (prime_age > 0)
                {
                    evacDrops.push_back({(long long)currentCity, (long long)prime_age, 0});
                    prime_age = 0;
                }
                break;
            }

            // Decide which shelter to go to (prioritize elderly if they have a shelter)
            int targetShelter = (nextShelterForElderly != -1) ? nextShelterForElderly : nextShelterForPrime;

            // Get path to the shelter
            const vector<int> &pathToShelter = shortestPaths[currentCity][targetShelter];

            // Skip first city if it's the current city
            int startIdx = (pathToShelter.size() > 0 && pathToShelter[0] == currentCity) ? 1 : 0;

            // Travel along the path to the shelter
            for (int j = startIdx; j < pathToShelter.size(); j++)
            {
                int nextCity = pathToShelter[j];

                // Calculate distance to next city
                int addedDistance = 0;
                for (const Road &road : adjacencyList[currentCity])
                {
                    if (road.to == nextCity)
                    {
                        addedDistance = road.length;
                        break;
                    }
                }

                // Check if elderly can reach the next city
                bool elderlyCanReach = (distanceTraveled + addedDistance) <= max_distance_elderly;

                // If elderly can't reach next city, drop them at current city
                if (elderly > 0 && !elderlyCanReach)
                {
                    evacDrops.push_back({(long long)currentCity, 0, (long long)elderly});
                    elderly = 0;
                }

                // Add next city to path
                evacPath.push_back(nextCity);

                // Update current city and distance
                currentCity = nextCity;
                distanceTraveled += addedDistance;

                // If current city is a shelter, try to drop people
                if (shelterCapacity.find(currentCity) != shelterCapacity.end() && shelterCapacity[currentCity] > 0)
                {
                    int capacity = shelterCapacity[currentCity];

                    // Drop elderly first (if they can reach)
                    int elderlyToDrop = min(elderly, capacity);
                    capacity -= elderlyToDrop;
                    elderly -= elderlyToDrop;

                    // Then drop prime age
                    int primeToDrop = min(prime_age, capacity);
                    capacity -= primeToDrop;
                    prime_age -= primeToDrop;

                    // Update shelter capacity
                    shelterCapacity[currentCity] -= (elderlyToDrop + primeToDrop);

                    // Record drop
                    if (elderlyToDrop > 0 || primeToDrop > 0)
                    {
                        evacDrops.push_back({(long long)currentCity, (long long)primeToDrop, (long long)elderlyToDrop});
                    }

                    // If all evacuees are dropped, break
                    if (prime_age == 0 && elderly == 0)
                    {
                        break;
                    }
                }
            }
        }

        // Set the path size
        path_size[i] = evacPath.size();

        // Allocate and copy the path
        paths[i] = new long long[evacPath.size()];
        for (int j = 0; j < evacPath.size(); j++)
        {
            paths[i][j] = evacPath[j];
        }

        // Set the number of drops
        num_drops[i] = evacDrops.size();

        // Allocate and copy the drops
        drops[i] = new long long *[evacDrops.size()];
        for (int j = 0; j < evacDrops.size(); j++)
        {
            drops[i][j] = new long long[3];
            drops[i][j][0] = evacDrops[j][0]; // City
            drops[i][j][1] = evacDrops[j][1]; // Prime age
            drops[i][j][2] = evacDrops[j][2]; // Elderly
        }
    }
}

int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        cerr << "Usage: " << argv[0] << " <input_file> <output_file>\n";
        return 1;
    }

    //--------------input--------------------------------
    ifstream infile(argv[1]); // Read input file from command-line argument
    if (!infile)
    {
        cerr << "Error: Cannot open file " << argv[1] << "\n";
        return 1;
    }

    long long num_cities;
    infile >> num_cities;
    cout << "num cities = " << num_cities << endl;

    long long num_roads;
    infile >> num_roads;
    cout << "num roads = " << num_roads << endl;

    // Store roads as a flat array: [u1, v1, length1, capacity1, u2, v2, length2, capacity2, ...]
    int *roads = new int[num_roads * 4];

    for (int i = 0; i < num_roads; i++)
    {
        infile >> roads[4 * i] >> roads[4 * i + 1] >> roads[4 * i + 2] >> roads[4 * i + 3];
    }

    int num_shelters;
    infile >> num_shelters;
    cout << "num shelters = " << num_shelters << endl;

    // Store shelters separately
    long long *shelter_city = new long long[num_shelters];
    long long *shelter_capacity = new long long[num_shelters];

    for (int i = 0; i < num_shelters; i++)
    {
        infile >> shelter_city[i] >> shelter_capacity[i];
    }

    int num_populated_cities;
    infile >> num_populated_cities;
    cout << "num pop cities = " << num_populated_cities << endl;

    // Store populated cities separately
    long long *city = new long long[num_populated_cities];
    long long *pop = new long long[num_populated_cities * 2]; // Flattened [prime-age, elderly] pairs

    for (long long i = 0; i < num_populated_cities; i++)
    {
        infile >> city[i] >> pop[2 * i] >> pop[2 * i + 1];
    }

    int max_distance_elderly;
    infile >> max_distance_elderly;
    cout << "max dist elderly= " << max_distance_elderly << endl;

    infile.close();

    //-------------------------end of input----------

    // Build adjacency list from roads array
    vector<vector<Road>> adjacencyList(num_cities);
    for (int i = 0; i < num_roads; i++)
    {
        int u = roads[4 * i];
        int v = roads[4 * i + 1];
        int length = roads[4 * i + 2];
        int capacity = roads[4 * i + 3];

        adjacencyList[u].push_back({v, length, capacity});
        adjacencyList[v].push_back({u, length, capacity});
    }

    // Create shelter capacity map and list
    unordered_map<int, int> shelterCapacity;
    vector<int> shelterCities;
    for (int i = 0; i < num_shelters; i++)
    {
        int city_id = shelter_city[i];
        int capacity = shelter_capacity[i];

        shelterCapacity[city_id] = capacity;
        shelterCities.push_back(city_id);
    }

    // Create populated city info map and list
    unordered_map<int, pair<int, int>> populatedCityInfo;
    vector<int> populatedCities;
    for (int i = 0; i < num_populated_cities; i++)
    {
        int city_id = city[i];
        int prime_age = pop[2 * i];
        int elderly = pop[2 * i + 1];

        populatedCityInfo[city_id] = {prime_age, elderly};
        populatedCities.push_back(city_id);
    }

    // Initialize distances and paths
    vector<vector<int>> distances(num_cities, vector<int>(num_cities, INT_MAX));
    vector<vector<vector<int>>> shortestPaths(num_cities, vector<vector<int>>(num_cities));

    // Compute shortest paths using CUDA
    computeShortestPathsCuda(adjacencyList, distances, shortestPaths, num_cities, roads, num_roads);

    // set your answer to these variables
    long long *path_size = new long long[num_populated_cities];
    long long **paths = new long long *[num_populated_cities];
    long long *num_drops = new long long[num_populated_cities];
    long long ***drops = new long long **[num_populated_cities];

    // Generate evacuation paths
    generateEvacuationPaths(
        num_cities, adjacencyList, populatedCities, populatedCityInfo,
        shelterCities, shelterCapacity, distances, shortestPaths,
        max_distance_elderly, path_size, paths, num_drops, drops, num_populated_cities);

    //------------output-----------------

    ofstream outfile(argv[2]); // Output file from command-line argument
    if (!outfile)
    {
        cerr << "Error: Cannot open file " << argv[2] << "\n";
        return 1;
    }

    for (long long i = 0; i < num_populated_cities; i++)
    {
        long long currentPathSize = path_size[i];
        for (long long j = 0; j < currentPathSize; j++)
        {
            outfile << paths[i][j] << " ";
        }
        outfile << "\n";
    }

    for (long long i = 0; i < num_populated_cities; i++)
    {
        long long currentDropSize = num_drops[i];
        for (long long j = 0; j < currentDropSize; j++)
        {
            for (int k = 0; k < 3; k++)
            {
                outfile << drops[i][j][k] << " ";
            }
        }
        outfile << "\n";
    }

    // Free allocated memory
    delete[] roads;
    delete[] shelter_city;
    delete[] shelter_capacity;
    delete[] city;
    delete[] pop;

    for (int i = 0; i < num_populated_cities; i++)
    {
        delete[] paths[i];
        for (int j = 0; j < num_drops[i]; j++)
        {
            delete[] drops[i][j];
        }
        delete[] drops[i];
    }

    delete[] path_size;
    delete[] paths;
    delete[] num_drops;
    delete[] drops;

    return 0;
}