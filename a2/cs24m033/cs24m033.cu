//--------changed <cuda/cuda_runtime.h> to <cuda_runtime.h> ---------
#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void dkernel(long int *matrix, long int *filter, long int *result, int h, int w, int c, int r, int s, int k)
{
    // get global thread position
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx % w;
    int y = idx / w;

    // storing the filter in shared memory
    extern __shared__ long int shared_filter[];

    // parallel storing the shared memory
    int filterSize = r * s * c * k;
    int threadsPerBlock = blockDim.x;
    for (int i = threadIdx.x; i < filterSize; i += threadsPerBlock)
    {
        shared_filter[i] = filter[i];
    }

    __syncthreads();

    // convulation code logic
    if (idx < h * w)
    {
        // loop through number of filters
        for (int f = 0; f < k; ++f)
        {
            long int sum = 0;

            // loop through channels
            for (int ch = 0; ch < c; ++ch)
            {
                for (int i = 0; i < r; ++i)
                {
                    for (int j = 0; j < s; ++j)
                    {
                        // storing the row & col of input matrix
                        int row = y + i - r / 2;
                        int col = x + j - s / 2;

                        // checking valid indices
                        if (row >= 0 && row < h && col >= 0 && col < w)
                        {
                            // applying the filter from shared memory to the matirx
                            sum += matrix[(ch * h + row) * w + col] *
                                   shared_filter[f * (r * s * c) + ch * (r * s) + i * s + j];
                        }
                    }
                }
            }
            // store final result
            result[f * h * w + idx] = sum;
        }
    }
}

int main(int argc, char **argv)
{
    int h, w, c;
    cin >> h >> w >> c;
    long int *h_mat = new long int[h * w * c];
    for (long int i = 0; i < h * w * c; i++)
    {
        cin >> h_mat[i];
    }

    int cf, r, s, k;
    cin >> cf >> r >> s >> k;

    long int *h_filter = new long int[r * s * c * k];
    for (long int i = 0; i < r * s * c * k; i++)
    {
        cin >> h_filter[i];
    }
    long int *h_ans = new long int[h * w * k];

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     **/
    /****************************************************Start Here***********************************************************/

    /**
        Do device allocations, kernel launches and copying everything here
        and the final answer should be stored back in h_ans, use cudaFree to free up the allocated memory on GPU
    */

    auto start = std::chrono::high_resolution_clock::now(); // keep it just before the kernel launch

    // allocate space in GPU
    long int *d_mat, *d_filter, *d_result;
    hipMalloc((void **)&d_mat, h * w * c * sizeof(long int));
    hipMalloc((void **)&d_filter, r * s * c * k * sizeof(long int));
    hipMalloc((void **)&d_result, h * w * k * sizeof(long int));

    // Copy data from host to device
    hipMemcpy(d_mat, h_mat, h * w * c * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, r * s * c * k * sizeof(long int), hipMemcpyHostToDevice);

    // deciding block size
    int blockSize = 256;
    int gridSize = (h * w + blockSize - 1) / blockSize;

    // sharedMemSize is the size of filter matrix
    int sharedMemSize = r * s * c * k * sizeof(long int);

    //----kernel launch-----
    dkernel<<<gridSize, blockSize, sharedMemSize>>>(d_mat, d_filter, d_result, h, w, c, r, s, k);
    //----------------------

    // copy results back to host
    hipMemcpy(h_ans, d_result, h * w * k * sizeof(long int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_mat);
    hipFree(d_filter);
    hipFree(d_result);

    auto end = std::chrono::high_resolution_clock::now(); // keep it just after the kernel launch
    std::chrono::duration<double> elapsed1 = end - start;

    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/

    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    hipDeviceSynchronize();
    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < h * k; i++)
        {
            for (long int j = 0; j < w; j++)
            {
                file << h_ans[i * w + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}
